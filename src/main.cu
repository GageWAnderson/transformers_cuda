#include <iostream>
#include <algorithm>
#include <string>
#include <vector>
#include "config.cuh"
#include "hip/hip_runtime.h"
#include "hipDNN.h"
#include "utils/utils.cuh"
#include "utils/softmax.cuh"
#include "embeddings/token_embeddings.cuh"
#include "embeddings/positional_encoding.cuh"
#include "tokenizer/vocab.cuh"
#include "tokenizer/tokenizer.cuh"
#include "decoder/decoder.cuh"
#include "hipblas.h"
#include "hiprand.h"
#include "layers/final_linear_layer.cuh"
#include "utils/debug.cuh"
#include "utils/load_weights.cuh"
#include "gpt2_weights.cuh"
#include <fstream>

// Function to display usage instructions
void printUsage();

// Helper function to load configuration
/**
 * @brief Loads configuration from file
 * @param config Reference to Config object
 * @return true if loaded successfully, false otherwise
 *
 * Attempts to load configuration from config.ini file.
 * Falls back to default values if load fails.
 */
bool loadConfiguration(Config &config)
{
    if (!config.loadFromFile("config/config.ini"))
    {
        debugPrint("Proceeding with default configuration values.\n");
        return false;
    }
    return true;
}

// Helper function to parse command-line arguments
/**
 * @brief Parses command line arguments
 * @param argc Argument count
 * @param argv Argument values
 * @param config Reference to Config object
 * @return true if parsed successfully, false if help requested or error
 *
 * Processes command line arguments to override config values.
 */
bool parseArguments(int argc, char *argv[], Config &config, std::string &weights_file)
{
    for (int i = 1; i < argc; ++i)
    {
        std::string arg = argv[i];
        if (arg.find("--weights=") == 0)
        {
            weights_file = arg.substr(10);
        }
        else if (arg == "--help" || arg == "-h")
        {
            printUsage();
            return false;
        }
        else
        {
            std::cerr << "Unknown argument: " << arg << std::endl;
            printUsage();
            return false;
        }
    }

    return true;
}

// Helper function to load vocabulary
/**
 * @brief Loads and displays vocabulary
 * @param vocab_file Path to vocabulary file
 * @param vocabulary Vector to store vocabulary
 *
 * Loads vocabulary from file and prints size information.
 */
void loadAndDisplayVocabulary(const std::string &vocab_file, std::vector<std::string> &vocabulary)
{
    loadVocabulary(vocab_file, vocabulary);
    debugPrint("Loaded vocabulary with %zu tokens.\n", vocabulary.size());
}

// Helper function to initialize cuDNN
/**
 * @brief Initializes cuDNN library
 * @return Initialized cuDNN handle
 *
 * Creates and returns cuDNN handle for use with neural network operations.
 */
hipdnnHandle_t initializeCUDNN()
{
    hipdnnHandle_t cudnn;
    checkCUDNN(hipdnnCreate(&cudnn));
    return cudnn;
}

// Helper function to run CLI server loop
/**
 * @brief Runs interactive CLI server
 * @param vocabulary Model vocabulary
 * @param d_token_embeddings Token embedding matrix
 * @param d_positional_encoding Positional encoding matrix
 * @param config Model configuration
 * @param decoder Decoder object
 * @param final_linear_layer FinalLinearLayer object
 * @param cudnn cuDNN handle
 * @param cublas cuBLAS handle
 *
 * Runs interactive command line interface for model inference.
 */
void runCLIServer(
    const std::vector<std::string> &vocabulary,
    float *d_token_embeddings,
    float *d_positional_encoding,
    const Config &config,
    Decoder &decoder,
    FinalLinearLayer &final_linear_layer,
    hipdnnHandle_t cudnn,
    hipblasHandle_t cublas)
{
    std::cout << "Transformer CLI server is running. Type 'exit' to quit.\n";

    // Allocate memory for decoder input and output
    float *d_decoder_input = nullptr;
    float *d_decoder_output = nullptr;
    size_t decoder_input_size = config.batch_size * config.hidden_dim * sizeof(float);
    hipMalloc(&d_decoder_input, decoder_input_size);
    hipMalloc(&d_decoder_output, decoder_input_size);

    // Create CUDA stream
    hipStream_t stream;
    hipStreamCreate(&stream);

    // Allocate memory for the current token embedding
    float *d_current_token_embedding = nullptr;
    hipMalloc(&d_current_token_embedding, decoder_input_size);

    std::string input;
    while (true)
    {
        std::cout << "\n> ";
        std::getline(std::cin, input);

        if (input == "exit")
        {
            break;
        }

        // Reset generation variables for new input
        std::vector<int> generated_tokens;
        int current_token_id = config.start_token_id;
        int generation_step = 0;

        debugPrint("\nGenerating tokens for input: %s\n", input.c_str());
        int seq_len = 1; // Sequence length is 1 for autoregressive decoding

        while (generation_step < config.max_generation_length)
        {
            // Get the embedding for the current token
            getTokenEmbedding(current_token_id, d_token_embeddings, d_current_token_embedding, config);

            // Prepare decoder input
            hipMemcpy(d_decoder_input, d_current_token_embedding, decoder_input_size, hipMemcpyDeviceToDevice);

            // Run decoder
            decoder.forward(d_decoder_output, d_decoder_input, nullptr, config.batch_size, seq_len, stream);

            // Allocate memory for logits
            float *d_logits = nullptr;
            size_t logits_size = config.batch_size * seq_len * config.vocab_size * sizeof(float);
            hipMalloc(&d_logits, logits_size);

            // Run final linear layer with token embeddings
            final_linear_layer.forward(d_decoder_output, d_logits, 1, d_token_embeddings);

            // Copy logits to host
            std::vector<float> h_logits(config.batch_size * seq_len * config.vocab_size);
            hipMemcpy(h_logits.data(), d_logits, logits_size, hipMemcpyDeviceToHost);

            // Select next token
            auto max_iter = std::max_element(h_logits.begin(), h_logits.end());
            int next_token_id = std::distance(h_logits.begin(), max_iter);

            // Append the token to generated sequence
            generated_tokens.push_back(next_token_id);

            // Check for stop token
            if (next_token_id == config.stop_token_id)
            {
                break;
            }

            // Update current token for next iteration
            current_token_id = next_token_id;
            generation_step++;

            // Cleanup
            hipFree(d_logits);
        }

        // Print the generated tokens all at once at the end of the generation
        for (int token_id : generated_tokens)
        {
            std::cout << vocabulary[token_id];
        }
        std::cout << std::endl; // New line after generation
    }

    // Cleanup
    hipStreamSynchronize(stream);
    hipStreamDestroy(stream);
    hipFree(d_decoder_input);
    hipFree(d_decoder_output);
    hipFree(d_current_token_embedding);
}

int main(int argc, char *argv[])
{
    Config config;

    // Load configurations from the config file
    loadConfiguration(config);

    // Parse command-line arguments and load weights
    std::string weights_file;
    if (!parseArguments(argc, argv, config, weights_file))
    {
        return 1;
    }

    // Load the vocabulary using the path from the config
    std::vector<std::string> vocabulary;
    loadAndDisplayVocabulary(config.vocab_file, vocabulary);

    // Initialize cuDNN
    hipdnnHandle_t cudnn = initializeCUDNN();

    // Create cuBLAS handle
    hipblasHandle_t cublas;
    hipblasCreate(&cublas);

    // Create cuRAND generator
    hiprandGenerator_t curand_gen;
    hiprandCreateGenerator(&curand_gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(curand_gen, 1234ULL);

    // Create token embeddings
    float *d_token_embeddings = nullptr;
    createTokenEmbeddings(config, &d_token_embeddings);

    // Create positional encodings
    float *d_positional_encoding = nullptr;
    createPositionalEncoding(config.max_seq_len, config.embedding_dim, &d_positional_encoding);

    // Print the positional encoding
    debugPrint("Positional encoding created with dimensions: %d x %d\n",
               config.max_seq_len, config.embedding_dim);

    // If weights file was specified, check architecture and try to load it
    GPT2Weights *weights = nullptr;
    if (!weights_file.empty())
    {
        if (config.model_arch == ModelArchitecture::GPT2)
        {
            weights = loadGPT2ModelWeights(weights_file);
            if (!weights)
            {
                std::cerr << "Failed to load weights from: " << weights_file << std::endl;
                return 1;
            }

            // Add validation here
            try
            {
                validate_weights(weights, config);
                debugPrint("Successfully loaded and validated GPT-2 model weights\n");
            }
            catch (const std::exception &e)
            {
                std::cerr << "Weight validation failed: " << e.what() << std::endl;
                delete weights;
                return 1;
            }
        }
        else
        {
            std::cerr << "Error: Model architecture is not supported. Cannot load weights." << std::endl;
            return 1;
        }
    }

    debugPrint("Weights loaded successfully, loading decoder\n");
    // Initialize Decoder with weights
    Decoder decoder(config, weights);

    // Create and initialize the FinalLinearLayer with weights
    debugPrint("Initializing FinalLinearLayer\n");
    FinalLinearLayer final_linear_layer(config, cublas, cudnn, weights);

    // Run the CLI server with all necessary components
    runCLIServer(vocabulary,
                 d_token_embeddings,
                 d_positional_encoding,
                 config,
                 decoder,
                 final_linear_layer,
                 cudnn,
                 cublas);

    // Cleanup token embeddings and positional encodings
    hipFree(d_token_embeddings);
    hipFree(d_positional_encoding);

    // Destroy cuBLAS handle
    hipblasDestroy(cublas);

    // Destroy cuRAND generator
    hiprandDestroyGenerator(curand_gen);

    // Destroy cuDNN handle
    hipdnnDestroy(cudnn);

    return 0;
}

void printUsage()
{
    std::cout << "Usage: transformer [options]\n";
    std::cout << "Options:\n";
    std::cout << "  --weights=FILE    Load model weights from SafeTensors file\n";
    std::cout << "  --help, -h        Show this help message\n";
}
