#include "decoder/decoder.cuh"
#include "layers/layer_norm.cuh"
#include "utils/utils.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

Decoder::Decoder(const Config &config)
{
    num_layers = config.num_layers;
    hidden_dim = config.hidden_dim;
    num_heads = config.num_heads;
    intermediate_dim = config.intermediate_dim;

    // Allocate arrays for each layer's components
    self_attention_layers = new MultiHeadAttention *[num_layers];
    encoder_attention_layers = new MultiHeadAttention *[num_layers];
    feed_forward_layers = new FeedForward *[num_layers];
    layer_norm1_layers = new LayerNorm *[num_layers];
    layer_norm2_layers = new LayerNorm *[num_layers];
    layer_norm3_layers = new LayerNorm *[num_layers];

    // Initialize components for each layer
    for (int i = 0; i < num_layers; ++i)
    {
        self_attention_layers[i] = new MultiHeadAttention(hidden_dim, num_heads);
        encoder_attention_layers[i] = new MultiHeadAttention(hidden_dim, num_heads);
        feed_forward_layers[i] = new FeedForward(hidden_dim, intermediate_dim);
        layer_norm1_layers[i] = new LayerNorm(hidden_dim);
        layer_norm2_layers[i] = new LayerNorm(hidden_dim);
        layer_norm3_layers[i] = new LayerNorm(hidden_dim);
    }
}

Decoder::~Decoder()
{
    // Delete components of each layer
    for (int i = 0; i < num_layers; ++i)
    {
        delete self_attention_layers[i];
        delete encoder_attention_layers[i];
        delete feed_forward_layers[i];
        delete layer_norm1_layers[i];
        delete layer_norm2_layers[i];
        delete layer_norm3_layers[i];
    }
    delete[] self_attention_layers;
    delete[] encoder_attention_layers;
    delete[] feed_forward_layers;
    delete[] layer_norm1_layers;
    delete[] layer_norm2_layers;
    delete[] layer_norm3_layers;
}

void Decoder::forward(float *output,
                      const float *input,
                      const float *encoder_output,
                      int batch_size,
                      int seq_len,
                      hipStream_t stream)
{
    // Allocate memory for intermediate outputs
    float *current_input = nullptr;
    float *current_output = nullptr;
    float *residual = nullptr;
    hipMalloc(&current_input, batch_size * seq_len * hidden_dim * sizeof(float));
    hipMalloc(&current_output, batch_size * seq_len * hidden_dim * sizeof(float));
    hipMalloc(&residual, batch_size * seq_len * hidden_dim * sizeof(float));

    // Copy input to current_input
    hipMemcpy(current_input, input, batch_size * seq_len * hidden_dim * sizeof(float), hipMemcpyDeviceToDevice);

    for (int i = 0; i < num_layers; ++i)
    {
        // Store the current input as residual
        hipMemcpy(residual, current_input, batch_size * seq_len * hidden_dim * sizeof(float), hipMemcpyDeviceToDevice);

        // Layer Norm 1
        layer_norm1_layers[i]->forward(current_input, current_input, seq_len, stream);

        // Masked Self-Attention
        // Note: For masked self-attention, you need to apply a mask to prevent attending to future positions.
        self_attention_layers[i]->forward(current_output, current_input, batch_size, seq_len, stream, /*mask=*/true);

        // Add & Norm
        add_tensors(current_output, residual, current_output, batch_size * seq_len * hidden_dim, stream);

        // Prepare residual for next sublayer
        hipMemcpy(residual, current_output, batch_size * seq_len * hidden_dim * sizeof(float), hipMemcpyDeviceToDevice);

        // Layer Norm 2
        layer_norm2_layers[i]->forward(current_output, current_output, seq_len, stream);

        // Encoder-Decoder Attention
        // Query comes from the previous sublayer's output, Key and Value come from the encoder output
        encoder_attention_layers[i]->forward(current_output, current_output, encoder_output, batch_size, seq_len, stream);

        // Add & Norm
        add_tensors(current_output, residual, current_output, batch_size * seq_len * hidden_dim, stream);

        // Prepare residual for next sublayer
        hipMemcpy(residual, current_output, batch_size * seq_len * hidden_dim * sizeof(float), hipMemcpyDeviceToDevice);

        // Layer Norm 3
        layer_norm3_layers[i]->forward(current_output, current_output, seq_len, stream);

        // Feed Forward
        feed_forward_layers[i]->forward(current_output, current_output, seq_len, stream);

        // Add & Norm
        add_tensors(current_output, residual, current_output, batch_size * seq_len * hidden_dim, stream);

        // Swap pointers for next layer
        std::swap(current_input, current_output);
    }

    // Copy the final output
    hipMemcpy(output, current_input, batch_size * seq_len * hidden_dim * sizeof(float), hipMemcpyDeviceToDevice);

    // Free intermediate memory
    hipFree(current_input);
    hipFree(current_output);
    hipFree(residual);
}
