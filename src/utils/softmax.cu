#include "utils/softmax.cuh"
#include "utils/utils.cuh"

void applySoftmax(hipdnnHandle_t &cudnn, float *d_input, float *d_output, int test_size)
{
    // Create tensor descriptor
    hipdnnTensorDescriptor_t input_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(
        input_descriptor,
        HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT,
        1, 1, 1, test_size));

    // Apply softmax
    float alpha = 1.0f, beta = 0.0f;
    checkCUDNN(hipdnnSoftmaxForward(
        cudnn,
        HIPDNN_SOFTMAX_ACCURATE,
        HIPDNN_SOFTMAX_MODE_INSTANCE,
        &alpha,
        input_descriptor,
        d_input,
        &beta,
        input_descriptor,
        d_output));

    // Cleanup
    hipdnnDestroyTensorDescriptor(input_descriptor);
}
