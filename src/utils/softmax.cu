#include "utils/softmax.cuh"
#include "utils/utils.cuh"

void applySoftmax(hipdnnHandle_t &cudnn, float *d_input, float *d_output, int batch_size, int num_classes)
{
    // Create tensor descriptor
    hipdnnTensorDescriptor_t tensorDesc;
    hipdnnCreateTensorDescriptor(&tensorDesc);
    hipdnnSetTensor4dDescriptor(tensorDesc,
                               HIPDNN_TENSOR_NCHW,
                               HIPDNN_DATA_FLOAT,
                               batch_size,   // N
                               num_classes,  // C
                               1,            // H
                               1);           // W

    // Apply softmax
    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipdnnSoftmaxForward(cudnn,
                        HIPDNN_SOFTMAX_ACCURATE,
                        HIPDNN_SOFTMAX_MODE_CHANNEL,
                        &alpha,
                        tensorDesc,
                        d_input,
                        &beta,
                        tensorDesc,
                        d_output);

    // Destroy tensor descriptor
    hipdnnDestroyTensorDescriptor(tensorDesc);
}
