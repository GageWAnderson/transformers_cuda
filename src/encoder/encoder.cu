#include "hip/hip_runtime.h"
#include "encoder/encoder.cuh"
#include "layers/layer_norm.cuh"
#include "utils/utils.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

/**
 * @brief Constructs an Encoder with the given configuration
 * @param config Configuration object containing model parameters
 *
 * Initializes a transformer encoder with the specified number of layers,
 * hidden dimensions, attention heads, and intermediate dimensions.
 * Allocates memory for all layer components including self-attention,
 * feed-forward networks, and layer normalization.
 */
Encoder::Encoder(const Config &config)
{
    num_layers = config.num_layers;
    hidden_dim = config.hidden_dim;
    num_heads = config.num_heads;
    intermediate_dim = config.intermediate_dim;

    // Allocate arrays for each layer's components
    self_attention_layers = new MultiHeadAttention *[num_layers];
    feed_forward_layers = new FeedForward *[num_layers];
    layer_norm1_layers = new LayerNorm *[num_layers];
    layer_norm2_layers = new LayerNorm *[num_layers];

    // Initialize components for each layer
    for (int i = 0; i < num_layers; ++i)
    {
        // Initialize weights and biases as null
        float *W_q_ptr = nullptr, *W_k_ptr = nullptr, *W_v_ptr = nullptr, *W_o_ptr = nullptr;
        float *b_q_ptr = nullptr, *b_k_ptr = nullptr, *b_v_ptr = nullptr, *b_o_ptr = nullptr;
        float *W1_ptr = nullptr, *b1_ptr = nullptr, *W2_ptr = nullptr, *b2_ptr = nullptr;

        self_attention_layers[i] = new MultiHeadAttention(hidden_dim, num_heads, W_q_ptr, W_k_ptr, W_v_ptr, W_o_ptr);
        feed_forward_layers[i] = new FeedForward(hidden_dim, intermediate_dim, W1_ptr, b1_ptr, W2_ptr, b2_ptr);
        layer_norm1_layers[i] = new LayerNorm(hidden_dim);
        layer_norm2_layers[i] = new LayerNorm(hidden_dim);
    }
}

/**
 * @brief Destructor for the Encoder class
 *
 * Cleans up all allocated memory for layer components including
 * self-attention layers, feed-forward networks, and layer normalization.
 */
Encoder::~Encoder()
{
    // Delete components of each layer
    for (int i = 0; i < num_layers; ++i)
    {
        delete self_attention_layers[i];
        delete feed_forward_layers[i];
        delete layer_norm1_layers[i];
        delete layer_norm2_layers[i];
    }
    delete[] self_attention_layers;
    delete[] feed_forward_layers;
    delete[] layer_norm1_layers;
    delete[] layer_norm2_layers;
}

/**
 * @brief Performs forward pass through the encoder
 * @param output Pointer to output tensor on device
 * @param input Pointer to input tensor on device
 * @param batch_size Number of sequences in batch
 * @param seq_len Length of input sequences
 * @param stream CUDA stream for asynchronous execution
 *
 * Processes input through multiple encoder layers with self-attention,
 * feed-forward networks, residual connections and layer normalization.
 * Uses multiple CUDA streams for parallel operations where possible.
 */
void Encoder::forward(float *output, const float *input, int batch_size, int seq_len, hipStream_t stream)
{
    // Allocate memory for intermediate outputs (update sizes to account for batch dimension)
    float *current_input = nullptr;
    float *current_output = nullptr;
    float *residual = nullptr;
    hipMalloc(&current_input, batch_size * seq_len * hidden_dim * sizeof(float));
    hipMalloc(&current_output, batch_size * seq_len * hidden_dim * sizeof(float));
    hipMalloc(&residual, batch_size * seq_len * hidden_dim * sizeof(float));
    hipMemcpy(current_input, input, batch_size * seq_len * hidden_dim * sizeof(float), hipMemcpyDeviceToDevice);

    for (int i = 0; i < num_layers; ++i)
    {
        // Store the current input as residual
        hipMemcpy(residual, current_input, batch_size * seq_len * hidden_dim * sizeof(float), hipMemcpyDeviceToDevice);

        // Create separate streams for parallel operations within the layer
        hipStream_t norm_stream, attn_stream;
        hipStreamCreate(&norm_stream);
        hipStreamCreate(&attn_stream);

        // Layer Norm 1 and Self-Attention can start in parallel
        layer_norm1_layers[i]->forward(current_output, current_input, seq_len, norm_stream);

        // Wait for norm to complete before attention
        hipStreamSynchronize(norm_stream);
        self_attention_layers[i]->forward(current_output, current_output, batch_size, seq_len, attn_stream);

        // Add & Norm
        hipStreamSynchronize(attn_stream);
        add_tensors(current_output, residual, current_output, batch_size * seq_len * hidden_dim, stream);

        // Layer Norm 2
        layer_norm2_layers[i]->forward(current_output, current_output, seq_len, stream);

        // Feed Forward
        feed_forward_layers[i]->forward(current_output, current_output, seq_len, stream);

        // Add & Prepare for next layer
        add_tensors(current_output, residual, current_output, batch_size * seq_len * hidden_dim, stream);

        // Clean up streams
        hipStreamDestroy(norm_stream);
        hipStreamDestroy(attn_stream);

        // Swap pointers for next layer
        std::swap(current_input, current_output);
    }

    // Copy the final output
    hipMemcpy(output, current_input, batch_size * seq_len * hidden_dim * sizeof(float), hipMemcpyDeviceToDevice);

    // Free intermediate memory
    hipFree(current_input);
    hipFree(current_output);
    hipFree(residual);
}
