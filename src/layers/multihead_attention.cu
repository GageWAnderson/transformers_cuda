#include "hip/hip_runtime.h"
#include "layers/multihead_attention.cuh"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <cassert>
#include <hipDNN.h>
#include "utils/softmax.cuh"

MultiHeadAttention::MultiHeadAttention(int hidden_dim, int num_heads)
{
    this->hidden_dim = hidden_dim;
    this->num_heads = num_heads;
    this->head_dim = hidden_dim / num_heads;

    // Initialize cuBLAS handle
    hipblasCreate(&cublas_handle);

    // Allocate memory for weights and biases
    size_t weight_size = hidden_dim * hidden_dim * sizeof(float); // Assuming square matrices for simplicity
    hipMalloc((void **)&W_q, weight_size);
    hipMalloc((void **)&W_k, weight_size);
    hipMalloc((void **)&W_v, weight_size);
    hipMalloc((void **)&W_o, weight_size);

    // Optionally allocate biases here if you use them

    // Initialize weights with random values
    hiprandGenerator_t curand_gen;
    hiprandCreateGenerator(&curand_gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(curand_gen, 1234ULL);

    hiprandGenerateUniform(curand_gen, W_q, hidden_dim * hidden_dim);
    hiprandGenerateUniform(curand_gen, W_k, hidden_dim * hidden_dim);
    hiprandGenerateUniform(curand_gen, W_v, hidden_dim * hidden_dim);
    hiprandGenerateUniform(curand_gen, W_o, hidden_dim * hidden_dim);

    // Destroy cuRAND generator
    hiprandDestroyGenerator(curand_gen);
}

MultiHeadAttention::~MultiHeadAttention()
{
    // Free weights and biases
    hipFree(W_q);
    hipFree(W_k);
    hipFree(W_v);
    hipFree(W_o);

    // Destroy cuBLAS handle
    hipblasDestroy(cublas_handle);
}

void MultiHeadAttention::forward(float *output, const float *input, int batch_size, int seq_len, hipStream_t stream)
{
    // Set the cuBLAS stream
    hipblasSetStream(cublas_handle, stream);

    // Dimensions
    int embed_dim = hidden_dim;
    int head_dim = this->head_dim; // Corrected to use the member variable

    // Allocate memory for Q, K, V, and attention scores
    float *Q;
    float *K;
    float *V;
    hipMalloc((void **)&Q, batch_size * seq_len * embed_dim * sizeof(float));
    hipMalloc((void **)&K, batch_size * seq_len * embed_dim * sizeof(float));
    hipMalloc((void **)&V, batch_size * seq_len * embed_dim * sizeof(float));

    // Linear projections: Q = input * W_q
    const float alpha = 1.0f;
    const float beta = 0.0f;
    hipblasSgemm(
        cublas_handle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        embed_dim,
        batch_size * seq_len,
        embed_dim,
        &alpha,
        W_q,
        embed_dim,
        input,
        embed_dim,
        &beta,
        Q,
        embed_dim);

    // Repeat for K and V
    hipblasSgemm(
        cublas_handle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        embed_dim,
        batch_size * seq_len,
        embed_dim,
        &alpha,
        W_k,
        embed_dim,
        input,
        embed_dim,
        &beta,
        K,
        embed_dim);

    hipblasSgemm(
        cublas_handle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        embed_dim,
        batch_size * seq_len,
        embed_dim,
        &alpha,
        W_v,
        embed_dim,
        input,
        embed_dim,
        &beta,
        V,
        embed_dim);

    // Reshape Q, K, V if necessary
    // Skipping for simplicity

    // Compute scaled dot-product attention scores
    float *attention_scores;
    hipMalloc((void **)&attention_scores, batch_size * num_heads * seq_len * seq_len * sizeof(float));

    const float scale = 1.0f / sqrtf((float)head_dim);

    // Compute attention scores using hipblasSgemmStridedBatched
    hipblasSgemmStridedBatched(
        cublas_handle,
        HIPBLAS_OP_T,
        HIPBLAS_OP_N,
        seq_len,
        seq_len,
        head_dim,
        &scale,
        K,
        head_dim,
        seq_len * head_dim,
        Q,
        head_dim,
        seq_len * head_dim,
        &beta,
        attention_scores,
        seq_len,
        seq_len * seq_len,
        batch_size * num_heads);

    // Apply softmax to attention_scores
    // Initialize cuDNN handle
    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);
    hipdnnSetStream(cudnn, stream);

    // Apply softmax using the provided function
    int total_elements = batch_size * num_heads * seq_len * seq_len;
    applySoftmax(cudnn, attention_scores, attention_scores, total_elements);

    // Destroy cuDNN handle
    hipdnnDestroy(cudnn);

    // Compute attention output: attention_output = attention_scores * V
    float *attention_output;
    hipMalloc((void **)&attention_output, batch_size * seq_len * embed_dim * sizeof(float));

    hipblasSgemmStridedBatched(
        cublas_handle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        head_dim,
        seq_len,
        seq_len,
        &alpha,
        V,
        head_dim,
        seq_len * head_dim,
        attention_scores,
        seq_len,
        seq_len * seq_len,
        &beta,
        attention_output,
        head_dim,
        seq_len * head_dim,
        batch_size * num_heads);

    // Concatenate heads and project the output: output = attention_output * W_o
    hipblasSgemm(
        cublas_handle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        embed_dim,
        batch_size * seq_len,
        embed_dim,
        &alpha,
        W_o,
        embed_dim,
        attention_output,
        embed_dim,
        &beta,
        output,
        embed_dim);

    // Free allocated memory
    hipFree(Q);
    hipFree(K);
    hipFree(V);
    hipFree(attention_scores);
    hipFree(attention_output);
}