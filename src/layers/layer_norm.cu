#include "hip/hip_runtime.h"
#include "layers/layer_norm.cuh"
#include "utils/utils.cuh"
#include "utils/debug.cuh"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <stdio.h>
#include <hipcub/hipcub.hpp>

LayerNorm::LayerNorm(int hidden_dim) : hidden_dim(hidden_dim)
{
    // Verify we have a valid CUDA device
    int device_count;
    CUDA_CHECK(hipGetDeviceCount(&device_count));
    if (device_count == 0) {
        throw std::runtime_error("No CUDA devices available");
    }

    // Allocate and initialize gamma and beta (scale and shift parameters)
    CUDA_CHECK(hipMalloc(&gamma, hidden_dim * sizeof(float)));
    CUDA_CHECK(hipMalloc(&beta, hidden_dim * sizeof(float)));

    // Initialize gamma to 1 and beta to 0
    float *h_gamma = (float *)malloc(hidden_dim * sizeof(float));
    float *h_beta = (float *)malloc(hidden_dim * sizeof(float));
    for (int i = 0; i < hidden_dim; ++i)
    {
        h_gamma[i] = 1.0f;
        h_beta[i] = 0.0f;
    }
    CUDA_CHECK(hipMemcpy(gamma, h_gamma, hidden_dim * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(beta, h_beta, hidden_dim * sizeof(float), hipMemcpyHostToDevice));

    free(h_gamma);
    free(h_beta);

    // Initialize cuBLAS
    hipblasCreate(&cublas_handle);
}

LayerNorm::~LayerNorm() noexcept
{
    // Free resources
    hipError_t err;
    err = hipFree(gamma);
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
    }
    err = hipFree(beta);
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
    }
    hipblasDestroy(cublas_handle);
}

__global__ void fused_layer_norm_kernel(
    const float* __restrict__ input,
    float* __restrict__ output,
    const float* __restrict__ gamma,
    const float* __restrict__ beta,
    int hidden_dim,
    int seq_len,
    float epsilon)
{
    extern __shared__ float shared_data[];
    
    int tid = threadIdx.x;
    int seq_idx = blockIdx.x;
    
    // Each block handles one sequence
    float sum = 0.0f;
    float sum_sq = 0.0f;
    
    // Cooperative loading and summation
    for (int i = tid; i < hidden_dim; i += blockDim.x) {
        float val = input[seq_idx * hidden_dim + i];
        sum += val;
        sum_sq += val * val;
    }
    
    // Block-level reduction
    __syncthreads();
    
    if (tid == 0) {
        float mean_val = sum / hidden_dim;
        float variance = (sum_sq / hidden_dim) - (mean_val * mean_val);
        shared_data[0] = mean_val;
        shared_data[1] = variance;
    }
    __syncthreads();
    
    // Normalize and apply gamma/beta
    float mean_val = shared_data[0];
    float variance = shared_data[1];
    
    for (int i = tid; i < hidden_dim; i += blockDim.x) {
        float val = input[seq_idx * hidden_dim + i];
        float norm_val = (val - mean_val) * rsqrtf(variance + epsilon);
        output[seq_idx * hidden_dim + i] = norm_val * gamma[i] + beta[i];
    }
}

void LayerNorm::forward(float* output, const float* input, int seq_len, hipStream_t stream) {
    // Check input parameters
    CUDA_CHECK(hipPeekAtLastError()); // Check for any previous errors
    
    const int BLOCK_SIZE = 256;
    const int shared_mem_size = 2 * sizeof(float); // For mean and variance
    
    // Validate pointers and parameters
    if (output == nullptr || input == nullptr || gamma == nullptr || beta == nullptr) {
        throw std::runtime_error("Null pointer passed to layer norm forward");
    }
    
    if (seq_len <= 0 || hidden_dim <= 0) {
        throw std::runtime_error("Invalid dimensions in layer norm forward");
    }
    
    // Calculate total elements to process (seq_len is actually batch_size * seq_len)
    int total_sequences = seq_len;
    
    fused_layer_norm_kernel<<<total_sequences, BLOCK_SIZE, shared_mem_size, stream>>>(
        input, output, gamma, beta, hidden_dim, total_sequences, 1e-5f);
    
    CUDA_CHECK(hipPeekAtLastError()); // Check for kernel launch errors
}

void LayerNorm::setGamma(float* gamma_weights) {
    CUDA_CHECK(hipMemcpy(gamma, gamma_weights, hidden_dim * sizeof(float), hipMemcpyDeviceToDevice));
}

void LayerNorm::setBeta(float* beta_weights) {
    CUDA_CHECK(hipMemcpy(beta, beta_weights, hidden_dim * sizeof(float), hipMemcpyDeviceToDevice));
}
