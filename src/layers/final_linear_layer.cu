#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include "layers/final_linear_layer.cuh"
#include "utils/softmax.cuh"
#include "utils/utils.cuh"
#include <hip/hip_runtime.h>

/**
 * @brief CUDA kernel for linear transformation
 * @param input Input tensor
 * @param weights Weight matrix
 * @param output Output tensor
 * @param vocab_size Size of vocabulary
 * @param batch_seq_len Combined batch and sequence length
 * @param hidden_dim Hidden dimension size
 *
 * Performs matrix multiplication between input and weights to produce logits.
 * Each thread computes one element of the output matrix.
 */
__global__ void linearTransformKernel(const float *input, const float *weights, float *output,
                                      int vocab_size, int batch_seq_len, int hidden_dim)
{
    // Calculate global thread indices
    int row = blockIdx.x * blockDim.x + threadIdx.x; // For vocab_size dimension
    int col = blockIdx.y * blockDim.y + threadIdx.y; // For batch_seq_len dimension

    if (row < vocab_size && col < batch_seq_len)
    {
        float sum = 0.0f;
        // Perform dot product between input and weights
        for (int k = 0; k < hidden_dim; k++)
        {
            sum += weights[row * hidden_dim + k] * input[col * hidden_dim + k];
        }
        output[col * vocab_size + row] = sum;
    }
}

/**
 * @brief Constructs final linear layer with configuration and weights
 * @param config Model configuration
 * @param cublas_handle cuBLAS handle
 * @param cudnn_handle cuDNN handle
 * @param weights GPT2Weights object containing model weights
 *
 * Initializes final linear layer and loads weights for projecting hidden states to vocabulary size.
 */
FinalLinearLayer::FinalLinearLayer(const Config &config,
                                   hipblasHandle_t &cublas_handle,
                                   hipdnnHandle_t &cudnn_handle,
                                   const GPT2Weights *weights)
    : config_(config), cublas_(cublas_handle), cudnn_(cudnn_handle)
{
    // Allocate memory for weights
    allocateWeights();

    if (weights)
    {
        // Calculate sizes
        size_t weights_size = config_.hidden_dim * config_.vocab_size * sizeof(float);
        size_t bias_size = config_.vocab_size * sizeof(float);

        // Copy weights to device
        hipMemcpy(d_linear_weights_, weights->getFinalLayerNormWeight(),
                   weights_size, hipMemcpyHostToDevice);

        // Copy bias if available
        float *bias = weights->getFinalLayerNormBias();
        if (bias)
        {
            hipMemcpy(d_linear_bias_, bias, bias_size, hipMemcpyHostToDevice);
        }
    }
}

/**
 * @brief Destructor for the FinalLinearLayer class
 *
 * Cleans up all allocated memory for layer components including
 * weights and layer normalization.
 */
FinalLinearLayer::~FinalLinearLayer()
{
    freeWeights();
}

/**
 * @brief Allocates memory for layer weights
 *
 * Allocates GPU memory for the weight matrix used in linear transformation.
 */
void FinalLinearLayer::allocateWeights()
{
    size_t weights_size = config_.hidden_dim * config_.vocab_size * sizeof(float);
    hipMalloc(&d_linear_weights_, weights_size);
}

/**
 * @brief Frees allocated weight memory
 *
 * Releases GPU memory used for weights when layer is destroyed.
 */
void FinalLinearLayer::freeWeights()
{
    if (d_linear_weights_)
    {
        hipFree(d_linear_weights_);
        d_linear_weights_ = nullptr;
    }

    if (d_linear_bias_)
    {
        hipFree(d_linear_bias_);
        d_linear_bias_ = nullptr;
    }
}

/**
 * @brief Performs forward pass through final linear layer
 * @param d_input Input hidden states
 * @param d_logits Output logits
 * @param seq_len Sequence length
 *
 * Projects hidden states to vocabulary size using linear transformation,
 * then applies softmax to get probability distribution over vocabulary.
 */
void FinalLinearLayer::forward(float *d_input, float *d_logits, int seq_len)
{
    // Dimensions for the linear layer
    int vocab_size = config_.vocab_size;
    int batch_seq_len = config_.batch_size * seq_len;
    int hidden_dim = config_.hidden_dim;

    // Define block and grid dimensions
    dim3 blockDim(16, 16); // 256 threads per block
    dim3 gridDim(
        (vocab_size + blockDim.x - 1) / blockDim.x,
        (batch_seq_len + blockDim.y - 1) / blockDim.y);

    std::cout << "Dimensions - vocab_size: " << vocab_size
              << ", batch_seq_len: " << batch_seq_len
              << ", hidden_dim: " << hidden_dim << std::endl;
    std::cout << "Grid dims - x: " << gridDim.x << ", y: " << gridDim.y << std::endl;

    // Launch custom linear transformation kernel
    linearTransformKernel<<<gridDim, blockDim>>>(
        d_input,
        d_linear_weights_,
        d_logits,
        vocab_size,
        batch_seq_len,
        hidden_dim);

    // Check for kernel launch errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess)
    {
        std::cerr << "CUDA error in linear transform kernel: " << hipGetErrorString(error) << std::endl;
    }

    // If bias is available, add it to the output
    if (d_linear_bias_)
    {
        // TODO: Add bias addition kernel call here
        // This would need to be implemented as a separate CUDA kernel
    }

    // Apply softmax to the logits
    applySoftmax(cudnn_, d_logits, d_logits, batch_seq_len, vocab_size);
}
