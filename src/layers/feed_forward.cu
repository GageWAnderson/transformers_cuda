#include "hip/hip_runtime.h"
#include "../../include/layers/feed_forward.cuh"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

// Activation function (ReLU)
__global__ void relu_activation(float *data, int size)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < size)
    {
        data[idx] = fmaxf(0.0f, data[idx]);
    }
}

// Kernel to add bias
__global__ void add_bias(float *data, const float *bias, int seq_len, int dim)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int total_size = seq_len * dim;
    if (idx < total_size)
    {
        int bias_idx = idx % dim;
        data[idx] += bias[bias_idx];
    }
}

// Helper function to initialize biases with zeros
void initialize_biases(float *d_biases, size_t size, hipStream_t stream)
{
    hipMemsetAsync(d_biases, 0, size, stream);
}

FeedForward::FeedForward(int hidden_dim, int intermediate_dim,
                         float *W1_ptr, float *b1_ptr,
                         float *W2_ptr, float *b2_ptr)
{
    this->hidden_dim = hidden_dim;
    this->intermediate_dim = intermediate_dim;

    // Initialize pointers - they will be set later via setters if null
    d_W1 = W1_ptr;
    d_b1 = b1_ptr;
    d_W2 = W2_ptr;
    d_b2 = b2_ptr;
}

FeedForward::~FeedForward()
{
}

void FeedForward::forward(float *output, const float *input, int seq_len, hipStream_t stream)
{
    // Implement the feed-forward network forward pass

    // Allocate intermediate memory
    float *d_intermediate = nullptr;
    size_t intermediate_size = seq_len * intermediate_dim * sizeof(float);
    hipMalloc(&d_intermediate, intermediate_size);

    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);
    hipblasSetStream(cublas_handle, stream);

    // Linear Layer 1: intermediate = input * W1 + b1
    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Compute input * W1
    // input: [seq_len, hidden_dim]
    // W1: [hidden_dim, intermediate_dim]
    // d_intermediate: [seq_len, intermediate_dim]
    hipblasSgemm(cublas_handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,
                intermediate_dim, seq_len, hidden_dim,
                &alpha,
                d_W1, intermediate_dim,
                input, hidden_dim,
                &beta,
                d_intermediate, intermediate_dim);

    // Add bias b1
    int threads = 256;
    int blocks = (seq_len * intermediate_dim + threads - 1) / threads;
    add_bias<<<blocks, threads, 0, stream>>>(d_intermediate, d_b1, seq_len, intermediate_dim);

    // Apply ReLU activation
    relu_activation<<<blocks, threads, 0, stream>>>(d_intermediate, seq_len * intermediate_dim);

    // Linear Layer 2: output = intermediate * W2 + b2
    // output: [seq_len, hidden_dim]
    hipblasSgemm(cublas_handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,
                hidden_dim, seq_len, intermediate_dim,
                &alpha,
                d_W2, hidden_dim,
                d_intermediate, intermediate_dim,
                &beta,
                output, hidden_dim);

    // Add bias b2
    blocks = (seq_len * hidden_dim + threads - 1) / threads;
    add_bias<<<blocks, threads, 0, stream>>>(output, d_b2, seq_len, hidden_dim);

    // Cleanup
    hipFree(d_intermediate);
    hipblasDestroy(cublas_handle);
}
